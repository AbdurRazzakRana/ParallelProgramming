#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <>
#include <hip/hip_runtime.h>

// CUDA kernel to count occurrences of the value 3 in the chunk of array
__global__ void count3s_kernel(const int* array, size_t size, int* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localCount = 0;

    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        if (array[i] == 3) {
            localCount++;
        }
    }

    atomicAdd(result, localCount);
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Error: Argument is not proper");
        return 1;
    }
    // Fetching the array length from argument
    int arraySize = atoi(argv[1]);

    clock_t start, end;
    double runtime;
    // timer starts
    start = clock();

    // accessting the file pointer to read
    FILE *fp;
    if ((fp = fopen("input.txt", "r")) == NULL) {
      fprintf(stderr, "Error: Unable to open the file.\n");
      return 1;
    }
    int x, actualCount=0, i;
    // reading the values from file and keeping them in host array.
    int* hostArray = new int[arraySize];
    for(i=0; i<arraySize; i++) {
        fscanf(fp,"%d", &x);
        hostArray[i] = x;
        if(x==3) actualCount++;
    }
    // allocating memory for the gpu threads and their local result 
    int* deviceArray;
    int* deviceResult;
    hipMalloc((void**)&deviceArray, arraySize * sizeof(int));
    hipMalloc((void**)&deviceResult, sizeof(int));

    // Copy array from host to device
    hipMemcpy(deviceArray, hostArray, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Initialize result on the device
    hipMemset(deviceResult, 0, sizeof(int));

    // finding the maximum block size in order to ensure maximum parallilism.
    int maxBlockSize, GridSize;
    hipOccupancyMaxPotentialBlockSize(&GridSize, &maxBlockSize, count3s_kernel, 0, 0);
    printf("Block used: %d\n", maxBlockSize);
    printf("Grid used: %d\n", GridSize);

    // Launch CUDA kernel to count occurrences of '3' in deviceArray
    count3s_kernel<<<GridSize, maxBlockSize>>>(deviceArray, arraySize, deviceResult);

    // Copy result back to host
    int hostResult;
    hipMemcpy(&hostResult, deviceResult, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    printf("Count from CUDA: %d\n", hostResult);
    
    // timer end
    end = clock();
    runtime = ((double) (end - start)) / CLOCKS_PER_SEC * 1000.0;
    printf("Total Execution time: %f ms\n", runtime);

    // Cleanup
    delete[] hostArray;
    hipFree(deviceArray);
    hipFree(deviceResult);
    return 0;
}